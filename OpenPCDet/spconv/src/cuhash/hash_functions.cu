#include "hip/hip_runtime.h"
#include <cassert>
#include <cuhash/debugging.h>
#include <cuhash/hash_functions.h>
#include <cuhash/hash_table.h>

namespace cuhash {

void GenerateFunctions(const unsigned N, const unsigned num_keys,
                       const unsigned *d_keys, const unsigned table_size,
                       uint2 *constants) {
  bool regenerate = true;

  while (regenerate) {
    regenerate = false;

    // Generate a set of hash function constants for this build attempt.
    for (unsigned i = 0; i < N; ++i) {
      // uint_distribution(random_engine) % kPrimeDivisor;
      // genrand_int32() % kPrimeDivisor;
      unsigned new_a = generate_random_uint32() % kPrimeDivisor;
      constants[i].x = (1 > new_a ? 1 : new_a);
      constants[i].y = generate_random_uint32() % kPrimeDivisor;
    }

#ifdef FORCEFULLY_GENERATE_NO_CYCLES
    // Ensure that every key gets N different slots.
    regenerate =
        CheckAssignedSameSlot(N, num_keys, d_keys, table_size, constants);
#endif
  }

#ifdef TAKE_HASH_FUNCTION_STATISTICS
  // Examine how well distributed the items are.
  TakeHashFunctionStatistics(num_keys, d_keys, table_size, constants, N);
#endif
}

}; // namespace cuhash
